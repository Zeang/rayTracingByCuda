#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/version.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include "../util/ray.h"
#include "../util/camera.h"
#include "../util/random.h"
#include "../util/util.h"
#include "../hitables/hitable_list.h"
#include "../hitables/sphere.h"
#include "../hitables/bvh.h"
#include "../materials/material.h"

using namespace std;

__device__ vec3 color(const ray& r, hitable** world, hiprandState* local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0f, 1.0f, 1.0f);
    vec3 attenuation = vec3(0, 0, 0);
    for (int i = 0; i < 50; ++i) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, cur_ray, local_rand_state)) {
                cur_attenuation *= attenuation;
            }
            else {
                return vec3(0, 0, 0);
            }
        }else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= max_x || j >= max_y) return;
    int pixel_index = j * max_x + i;
    // Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void render(vec3* fb, int max_x, int max_y, int ns, camera **cam, hitable ** world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; ++s) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    fb[pixel_index] = col / (float)ns;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable** d_list, hitable** d_world, camera** d_camera, int nx, int ny, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0, -1000.0, -1), 1000,
            new lambertian(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a + RND, 0.2, b + RND);
                if (choose_mat < 0.8f) {
                    d_list[i++] = new moving_sphere(
                        center,
                        center + vec3(0, 0.5*RND, 0),
                        0.0, 1.0, 0.2,
                        new lambertian(vec3(RND * RND, RND * RND, RND * RND)));
                }
                else if (choose_mat < 0.95f) {
                    d_list[i++] = new sphere(center, 0.2,
                        new metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        /**rand_state = local_rand_state;*/
        *d_world = new hitable_list(d_list, 22 * 22 + 1 + 3);
        printf("debug1\n");
        //*d_world = new bvh_node(d_list, 22 * 22 + 1 + 3, 0.0, 1.0, rand_state);
        printf("debug2\n");

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0; (lookfrom - lookat).length();
        float aperture = 0.1;
        *d_camera = new camera(lookfrom,
            lookat,
            vec3(0, 1, 0),
            30.0,
            float(nx) / float(ny),
            aperture,
            dist_to_focus,
            0.0f,
            1.0f);
    }
}

__global__ void free_world(hitable** d_list, hitable** d_world, camera** d_camera) {
    for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
        delete ((sphere*)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete* d_world;
    delete* d_camera;
}

int main(void)
{
    clock_t tStart, tEnd;
    int nx = 1200, ny = 800;
    int ns = 100;
    int tx = 8, ty = 8;
    ofstream outfile;
    outfile.open("pic/rayTracingMotionBlur2.ppm");
    outfile << "P3\n" << nx << " " << ny << "\n255\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    tStart = clock();
    // allocate FB
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init << <1, 1 >> > (d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make world of hitables
    hitable** d_list;
    int num_hitables = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void**)&d_list, num_hitables * sizeof(hitable*)));
    hitable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    create_world << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    tEnd = clock();
    cout << "The malloc cost " << (double)(tEnd - tStart) * 1000 / CLOCKS_PER_SEC << " ms." << endl;
    tStart = clock();

    dim3 blocks((nx - 1) / tx + 1, (ny - 1) / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render <<<blocks, threads >>> (fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    
    tEnd = clock();
    cout << "The render cost " << (double)(tEnd - tStart) * 1000 / CLOCKS_PER_SEC << " ms." << endl;
    tStart = clock();

    for (int j = ny - 1; j >= 0; --j) {
        for (int i = 0; i < nx; ++i) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * sqrt(fb[pixel_index].r()));
            int ig = int(255.99 * sqrt(fb[pixel_index].g()));
            int ib = int(255.99 * sqrt(fb[pixel_index].b()));
            outfile << ir << " " << ig << " " << ib << endl;
        }
    }
    tEnd = clock();
    cout << "The output cost " << (double)(tEnd - tStart) * 1000 / CLOCKS_PER_SEC << " ms." << endl;

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
    return 0;
}