#include "hip/hip_runtime.h"
#include "common.h"
#include "renderer.h"
#include "scene.cuh"
#include "window.h"
#include "camera.h"

#include "../hitables/sphere.h"
#include "../hitables/hitable_list.h"
#include "../materials/material.h"

CUDA_DEV int numHitables = 0;

#ifdef CUDA_ENABLED
void initializeWorldCuda(bool showWindow, bool writeImagePPM,
	bool writeImagePNG, hitable*** list, hitable** world, Window** w,
	Image** image, camera** cam, Renderer** renderer)
{
	int choice = 3;

	switch (choice)
	{
	case 0:
		numHitables = 4;
		break;
	case 1:
		numHitables = 58;
		break;
	case 2:
		numHitables = 901;
		break;
	case 3:
		numHitables = 102;
		break;
	case 4:
		numHitables = 68;
		break;
	}

	// World
	checkCudaErrors(hipMallocManaged(list, numHitables * sizeof(hitable*)));
	hitable** worldPtr;
	checkCudaErrors(hipMallocManaged(&worldPtr, sizeof(hitable*)));
	switch (choice)
	{
	case 0:
		simpleScene << <1, 1 >> > (*list, worldPtr);
		break;
	case 1:
		simpleScene2 << <1, 1 >> > (*list, worldPtr);
		break;
	case 2:
		randomScene << <1, 1 >> > (*list, worldPtr);
		break;
	case 3:
		randomScene2 << <1, 1 >> > (*list, worldPtr);
		break;
	case 4:
		randomScene3 << <1, 1 >> > (*list, worldPtr);
		break;
	}
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	*world = *worldPtr;
	checkCudaErrors(hipFree(worldPtr));

	// camera
	vec3 lookFrom(13.0f, 2.0f, 3.0f);
	vec3 lookAt(0.0f, 0.0f, 0.0f);
	checkCudaErrors(hipMallocManaged(cam, sizeof(camera)));
	new (*cam)camera(lookFrom, lookAt, vec3(0.0f, 1.0f, 0.0f),
		20.0f, float(nx) / float(ny), distToFocus);

	// Renderer
	checkCudaErrors(hipMallocManaged(renderer, sizeof(Renderer)));
	new(*renderer) Renderer(showWindow, writeImagePPM, writeImagePNG);

	// Image
	checkCudaErrors(hipMallocManaged(image, sizeof(Image)));
	new(*image) Image(showWindow, writeImagePPM || writeImagePNG,
		nx, ny, tx, ty);

	// Window
	if (showWindow)
		*w = new Window(*cam, *renderer, nx, ny, thetaInit, phiInit,
			zoomScale, stepScale);
}

CUDA_GLOBAL void freeWorldCuda(hitable** list, hitable** world)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		for (int i = 0; i < numHitables; ++i)
		{
			delete ((sphere*)list[i])->mat_ptr;
			delete list[i];
		}
		//delete* world;
	}
}

void destroyWorldCuda(bool showWindow, hitable** list, hitable* world, Window* w, Image* image, camera* cam, Renderer* render)
{
	freeWorldCuda << <1, 1 >> > (list, &world);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	// checkCudaErrors(hipFree(list));
	// checkCudaErrors(hipFree(world));
	//checkCudaErrors(hipFree(w));
	checkCudaErrors(hipFree(image));
	checkCudaErrors(hipFree(cam));
	checkCudaErrors(hipFree(render));

	hipDeviceReset();
}

CUDA_GLOBAL void render(camera* cam, Image* image, hitable* world,
	Renderer* render, int sampleCount)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if ((i >= image->nx) || (j >= image->ny))
		return;

	int pixelIndex = j * image->nx + i;

	for (int s = 0; s < nsBatch; ++s)
	{
		RandomGenerator rng(sampleCount * nsBatch + s, i * image->nx + j);
		float u = float(i + rng.get1f()) / float(image->nx);	// left to right
		float v = float(j + rng.get1f()) / float(image->ny);	// botton to top
		ray r = cam->get_ray(rng, u, v);

		image->pixels[pixelIndex] += render->color(rng, r, world, 0);
	}
	
	vec3 col = image->pixels[pixelIndex] / (sampleCount * nsBatch);
	image->pixels2[pixelIndex] = col;
}
#endif	// CUDA_ENABLED

CUDA_GLOBAL void display(Image* image)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int pixelIndex = j * image->nx + i;

	vec3 col = image->pixels2[pixelIndex];
	// Gamma encoding of images is used to optimize the usage of bits
	// when encoding an image, or bandwidth used to transport an image,
	// by taking advantage of the non-linear manner in which humans perceive
	// light and color. (wikipedia)

	// we use gamma 2: raising the color to the power 1/gamma (1/2)
	col = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));

	int ir = clamp(int(255.f * col[0]), 0, 255);
	int ig = clamp(int(255.99f * col[1]), 0, 255);
	int ib = clamp(int(255.99f * col[2]), 0, 255);

	if (image->writeImage)
	{
		// PNG
		int index = (image->ny - 1 - j) * image->nx + i;
		int index3 = 3 * index;

		image->fileOutputImage[index3 + 0] = ir;
		image->fileOutputImage[index3 + 1] = ig;
		image->fileOutputImage[index3 + 2] = ib;
	}

	if (image->showWindow)
		image->windowPixels[(image->ny - j - 1) * image->nx + i] = (ir << 16) | (ig << 8) | (ib);
}

#ifdef CUDA_ENABLED
void Renderer::cudaRender(uint32_t* windowPixels, camera* cam,
	hitable* world, Image* image, int sampleCount, uint8_t* fileOutputImage)
{
	dim3 blocks((image->nx + image->tx - 1) / image->tx, (image->ny + image->ty - 1) / image->ty);
	dim3 threads(image->tx, image->ty);

	// Kernel call for the computation of pixel colors.
	render << <blocks, threads >> > (cam, image, world, this, sampleCount);

	// Denoise here.
#ifdef OIDN_ENABLED
	checkCudaErrors(hipDeviceSynchronize());
	image->denoise();
	checkCudaErrors(hipDeviceSynchronize());
#endif	// OIDN_ENABLED
	// Kernel call to fill the output buffers.
	display << <blocks, threads >> > (image);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}
#endif	// CUDA_ENABLED
